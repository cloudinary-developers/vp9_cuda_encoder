#include "hip/hip_runtime.h"
/*
    Cuda accelerated motion estimation for VP8 libvpx encoder
    by Pietro Paglierani, Giuliano Grossi, Federico Pedersini and Alessandro Petrini

    for Italtel and Universita' degli Studi di Milano
    2015-2016, Milano
*/

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <wchar.h>
#include <locale.h>

#include "vpx_config.h"
#include "cuda/typedef_cuda.h"
#include "cuda/me_cuda.h"

#ifdef __cplusplus
extern "C" {
#endif
#if HAVE_CUDA_ENABLED_DEVICE

__device__ __constant__ MV MV_16x12_lookup_tex[] = {                      {-12,-2}, {-12, 0}, {-12, 2},		// Unit: pixel
		                                              {-10,-5}, {-10,-3}, {-10,-1}, {-10, 1}, {-10, 3}, {-10, 5},
		                                 {-8,-8},  {-8,-6},  {-8,-4},  {-8,-2},  {-8, 0},  {-8, 2},  {-8, 4},  {-8, 6},  {-8, 8},
		                           {-6,-9},  {-6,-7},  {-6,-5},  {-6,-3},  {-6,-1},  {-6, 1},  {-6, 3},  {-6, 5},  {-6, 7},  {-6, 9},
		           {-4,-12},  {-4,-10},  {-4,-8},  {-4,-6},  {-4,-4},  {-4,-2},  {-4, 0},  {-4, 2},  {-4, 4},  {-4, 6},  {-4, 8},  {-4,10},  {-4,12},
	        {-2,-13},   {-2,-11},  {-2,-9},  {-2,-7},  {-2,-5},  {-2,-3},  {-2,-1},  {-2, 1},  {-2, 3},  {-2, 5},  {-2, 7},  {-2, 9},  {-2,11},  {-2,13},
{0,-16},  {0,-14},  {0,-12},   {0,-10},   {0,-8},   {0,-6},   {0,-4},   {0,-2},   {0, 0},   {0, 2},   {0, 4},   {0, 6},   {0, 8},   {0,10},   {0,12},   {0,14},   {0,16},
		     {2,-13},    {2,-11},   {2,-9},   {2,-7},   {2,-5},   {2,-3},   {2,-1},   {2, 1},   {2, 3},   {2, 5},   {2, 7},   {2, 9},   {2,11},   {2,13},
		            {4,-12},   {4,-10},   {4,-8},   {4,-6},   {4,-4},   {4,-2},   {4, 0},   {4, 2},   {4, 4},   {4, 6},   {4, 8},   {4,10},   {4,12},
		                            {6,-9},   {6,-7},   {6,-5},   {6,-3},   {6,-1},   {6, 1},   {6, 3},   {6, 5},   {6, 7},   {6, 9},
		                                  {8,-8},   {8,-6},   {8,-4},   {8,-2},   {8, 0},   {8, 2},   {8, 4},   {8, 6},   {8, 8},
		                                               {10,-5},  {10,-3},   {10,-1},  {10, 1},  {10, 3},  {10, 5},
		                                                               {12,-2},  {12, 0},  {12, 2},
		{0, 0} }; // 127 + 1 candidati

__device__ __constant__ MV_ref MV_lookup_refin_tex[] = { // Unit: pixel
                                                                      {-1.75,-0.25},              {-1.75, 0.25},
                                                        {-1.50,-0.50},              {-1.50, 0.00},              {-1.50, 0.50},
                                          {-1.25,-0.75},              {-1.25,-0.25},              {-1.25, 0.25},              {-1.25, 0.75},
                            {-1.00,-1.00},              {-1.00,-0.50},              {-1.00, 0.00},              {-1.00, 0.50},              {-1.00, 1.00},
                   {-0.75,-1.25},         {-0.75,-0.75},              {-0.75,-0.25},{-0.75, 0.00},{-0.75, 0.25},              {-0.75, 0.75},          {-0.75, 1.25},
         {-0.50,-1.50},     {-0.50,-1.00},              {-0.50,-0.50},{-0.50,-0.25},{-0.50, 0.00},{-0.50, 0.25},{-0.50, 0.50},              {-0.50, 1.00},     {-0.50, 1.50},
{-0.25,-1.75},     {-0.25,-1.25},         {-0.25,-0.75},{-0.25,-0.50},{-0.25,-0.25},{-0.25, 0.00},{-0.25, 0.25},{-0.25, 0.50},{-0.25, 0.75},          {-0.25, 1.25},     {-0.25, 1.75},
         { 0.00,-1.50},     { 0.00,-1.00},{ 0.00,-0.75},{ 0.00,-0.50},{ 0.00,-0.25},{ 0.00, 0.00},{ 0.00, 0.25},{ 0.00, 0.50},{ 0.00, 0.75},{ 0.00, 1.00},     { 0.00, 1.50},
{ 0.25,-1.75},     { 0.25,-1.25},         { 0.25,-0.75},{ 0.25,-0.50},{ 0.25,-0.25},{ 0.25, 0.00},{ 0.25, 0.25},{ 0.25, 0.50},{ 0.25, 0.75},          { 0.25, 1.25},     { 0.25, 1.75},
         { 0.50,-1.50},     { 0.50,-1.00},              { 0.50,-0.50},{ 0.50,-0.25},{ 0.50, 0.00},{ 0.50, 0.25},{ 0.50, 0.50},              { 0.50, 1.00},     { 0.50, 1.50},
                   { 0.75,-1.25},         { 0.75,-0.75},              { 0.75,-0.25},{ 0.75, 0.00},{ 0.75, 0.25},              { 0.75, 0.75},          { 0.75, 1.25},
                            { 1.00,-1.00},              { 1.00,-0.50},              { 1.00, 0.00},              { 1.00, 0.50},              { 1.00, 1.00},
                                          { 1.25,-0.75},              { 1.25,-0.25},              { 1.25, 0.25},              { 1.25, 0.75},
                                                        { 1.50,-0.50},              { 1.50, 0.00},              { 1.50, 0.50},
                                                                      { 1.75,-0.25},              { 1.75, 0.25}
}; // 93 candidati

__inline__ __device__ uint32_t __vvariance4( uint32_t u, uint32_t v )
{
	uint32_t w = 0;
    asm volatile("{\
                    .reg .u32                   t1;\
                    vabsdiff4.u32.u32.u32.sat	t1, %1, %2, 0;\
                    vmad.u32.u32.u32.sat		%0, t1.b0, t1.b0, 0;\
                    vmad.u32.u32.u32.sat		%0, t1.b1, t1.b1, %0;\
                    vmad.u32.u32.u32.sat		%0, t1.b2, t1.b2, %0;\
                    vmad.u32.u32.u32.sat		%0, t1.b3, t1.b3, %0;}"  : "=r"(w) : "r"(u), "r"(v));
    return w;
}

__inline__ __device__ uint32_t __vabsdiff4( uint32_t u, uint32_t v )
{
	uint32_t w = 0;
	//asm volatile("vabsdiff4.u32.u32.u32.sat.add %0, %1, %2, %3;" : "=r"(w) : "r"(u), "r"(v), "r"(w));
	//asm volatile("vabsdiff4.u32.u32.u32.sat.add %0, %1, %2, 0;" : "=r"(w) : "r"(u), "r"(v));
	asm volatile("vabsdiff4.u32.u32.u32.add %0, %1, %2, %3;" : "=r"(w) : "r"(u), "r"(v), "r"(w)); //Solo compute 3.5
	return w;
}


__global__ void me_cuda_tex ( const hipTextureObject_t in_tex, const hipTextureObject_t ref_tex,
				int const streamID, int const streamSize, int const stride, int const width, int const num_MB_width, int const split_on,
				int_mv * __restrict__ const MVs_g, int_mv * __restrict__ const MVs_split_g ) {

	__shared__ uint32_t diff[128][32];
	__shared__ uint8_t minpos[32];
	__shared__ uint8_t minpos_refin[32];

	// configurazione di lancio: blocks per grid:   16 x 1 x 1
	//							 threads per block:  4 x 8 x 1

	int32_t TID = threadIdx.y * blockDim.x + threadIdx.x;	// Thread Index (0..32)
	int32_t i, j;

	int32_t MBoffset = streamID * streamSize + blockIdx.x;
	int32_t blockX = MBoffset % num_MB_width;		// colonna
	int32_t blockY = MBoffset / num_MB_width;		// riga
	// Occhio: immagine di riferimento ha cornice (larghezza tot = stride) mentre immagine input no (largh tot = width)
	float intex_offset_x  = 16 * blockX + 4 * threadIdx.x;
	float intex_offset_y  = 16 * blockY + 2 * threadIdx.y;
	float reftex_offset_x = intex_offset_x + 32.0;
	float reftex_offset_y = intex_offset_y + 32.0;

	float saved_reftex_x = reftex_offset_x;
	float saved_reftex_y = reftex_offset_y;

    MV_ref iter_mv = {0.0,0.0};
/*
	uint32_t img0 = (uint32_t) ( ( (uint8_t)(tex2D<float>( in_tex, intex_offset_x + 3.5, intex_offset_y + 0.5 ) * 255.0) << 24) |
								 ( (uint8_t)(tex2D<float>( in_tex, intex_offset_x + 2.5, intex_offset_y + 0.5 ) * 255.0) << 16) |
							     ( (uint8_t)(tex2D<float>( in_tex, intex_offset_x + 1.5, intex_offset_y + 0.5 ) * 255.0) <<  8) |
							     ( (uint8_t)(tex2D<float>( in_tex, intex_offset_x + 0.5, intex_offset_y + 0.5 ) * 255.0)      ) );
	uint32_t img1 = (uint32_t) ( ( (uint8_t)(tex2D<float>( in_tex, intex_offset_x + 3.5, intex_offset_y + 1.5 ) * 255.0) << 24) |
								 ( (uint8_t)(tex2D<float>( in_tex, intex_offset_x + 2.5, intex_offset_y + 1.5 ) * 255.0) << 16) |
							     ( (uint8_t)(tex2D<float>( in_tex, intex_offset_x + 1.5, intex_offset_y + 1.5 ) * 255.0) <<  8) |
							     ( (uint8_t)(tex2D<float>( in_tex, intex_offset_x + 0.5, intex_offset_y + 1.5 ) * 255.0)      ) );
*/
    uint32_t img0 = (uint32_t) ( ( (uint8_t)(tex2D<float>( in_tex, intex_offset_x + 3.5, intex_offset_y + 1.5 ) * 255.0) << 24) |
                                 ( (uint8_t)(tex2D<float>( in_tex, intex_offset_x + 2.5, intex_offset_y + 0.5 ) * 255.0) << 16) |
                                 ( (uint8_t)(tex2D<float>( in_tex, intex_offset_x + 1.5, intex_offset_y + 1.5 ) * 255.0) <<  8) |
                                 ( (uint8_t)(tex2D<float>( in_tex, intex_offset_x + 0.5, intex_offset_y + 0.5 ) * 255.0)      ) );

#if HAVE_CUDA_MV_ITER
    int8_t maxloops = 3;
    while (maxloops > 0) {
#endif
	/* Organizzazione dei thread all'interno del macroblocco. Ogni thread considera 4 pixel e i 4 immediatamente sottostanti.
		Accesso a memoria globale non e' ottimale (coalescenza a gruppi di quattro), ma questo schema permette di raggruppare
		le sad in somme parziali per calcolare tutte le splitmv.
	 ╔══════════╦══════════╦══════════╦══════════╗
	 ║  TID  0  ║  TID  1  ║  TID  2  ║  TID  3  ║
     ╟──────────╫──────────╫──────────╫──────────╢
     ║  TID  4  ║  TID  5  ║  TID  6  ║  TID  7  ║
     ╠══════════╬══════════╬══════════╬══════════╣
	 ║  TID  9  ║  TID  9  ║  TID 10  ║  TID 11  ║
     ╟──────────╫──────────╫──────────╫──────────╢
     ║  TID 12  ║  TID 13  ║  TID 14  ║  TID 15  ║
     ╠══════════╬══════════╬══════════╬══════════╣
	 ║  TID 16  ║  TID 17  ║  TID 18  ║  TID 19  ║
     ╟──────────╫──────────╫──────────╫──────────╢
     ║  TID 20  ║  TID 21  ║  TID 22  ║  TID 23  ║
     ╠══════════╬══════════╬══════════╬══════════╣
	 ║  TID 24  ║  TID 25  ║  TID 26  ║  TID 27  ║
     ╟──────────╫──────────╫──────────╫──────────╢
     ║  TID 28  ║  TID 29  ║  TID 30  ║  TID 31  ║
     ╚══════════╩══════════╩══════════╩══════════╝
	 */


	 /*
		Calcolo delle sad, risultati memorizzati nella matrice diff.
                32          32 TID = 32 sotto blocchi, ognuno contenente sad parziali
		 /             \
		┌───────────────┐
		│               │
		│               │
		│               │
		│ diff[128][32] │   128 candidati mv
		│               │
		│               │
		│               │
		└───────────────┘
		Ogni thread si fa carico si un sottoblocco di 8 pixel e calcola la sad per ogni
		candidato mv
	 */
	for (i = 0; i < 128; i++){
		int32_t sad_result;
		MV offset_mv = MV_16x12_lookup_tex[i];
        /*
		uint32_t ref0 = (uint32_t) ( ( (uint8_t)(tex2D<float>( ref_tex, reftex_offset_x + 3.5 + offset_mv.col + iter_mv.col, reftex_offset_y + 0.5 + offset_mv.row + iter_mv.row ) * 255) << 24) |
									 ( (uint8_t)(tex2D<float>( ref_tex, reftex_offset_x + 2.5 + offset_mv.col + iter_mv.col, reftex_offset_y + 0.5 + offset_mv.row + iter_mv.row ) * 255) << 16) |
								     ( (uint8_t)(tex2D<float>( ref_tex, reftex_offset_x + 1.5 + offset_mv.col + iter_mv.col, reftex_offset_y + 0.5 + offset_mv.row + iter_mv.row ) * 255) <<  8) |
								     ( (uint8_t)(tex2D<float>( ref_tex, reftex_offset_x + 0.5 + offset_mv.col + iter_mv.col, reftex_offset_y + 0.5 + offset_mv.row + iter_mv.row ) * 255)      ) );
		uint32_t ref1 = (uint32_t) ( ( (uint8_t)(tex2D<float>( ref_tex, reftex_offset_x + 3.5 + offset_mv.col + iter_mv.col, reftex_offset_y + 1.5 + offset_mv.row + iter_mv.row ) * 255) << 24) |
									 ( (uint8_t)(tex2D<float>( ref_tex, reftex_offset_x + 2.5 + offset_mv.col + iter_mv.col, reftex_offset_y + 1.5 + offset_mv.row + iter_mv.row ) * 255) << 16) |
								     ( (uint8_t)(tex2D<float>( ref_tex, reftex_offset_x + 1.5 + offset_mv.col + iter_mv.col, reftex_offset_y + 1.5 + offset_mv.row + iter_mv.row ) * 255) <<  8) |
								     ( (uint8_t)(tex2D<float>( ref_tex, reftex_offset_x + 0.5 + offset_mv.col + iter_mv.col, reftex_offset_y + 1.5 + offset_mv.row + iter_mv.row ) * 255)      ) );
		sad_result  = __vabsdiff4( img0, ref0 );
		sad_result += __vabsdiff4( img1, ref1 );
        */
		//sad_result  = __vvariance4( img0, ref0 );
		//sad_result += __vvariance4( img1, ref1 );
        uint32_t ref0 = (uint32_t) ( ( (uint8_t)(tex2D<float>( ref_tex, reftex_offset_x + 3.5 + offset_mv.col + iter_mv.col, reftex_offset_y + 1.5 + offset_mv.row + iter_mv.row ) * 255) << 24) |
									 ( (uint8_t)(tex2D<float>( ref_tex, reftex_offset_x + 2.5 + offset_mv.col + iter_mv.col, reftex_offset_y + 0.5 + offset_mv.row + iter_mv.row ) * 255) << 16) |
								     ( (uint8_t)(tex2D<float>( ref_tex, reftex_offset_x + 1.5 + offset_mv.col + iter_mv.col, reftex_offset_y + 1.5 + offset_mv.row + iter_mv.row ) * 255) <<  8) |
								     ( (uint8_t)(tex2D<float>( ref_tex, reftex_offset_x + 0.5 + offset_mv.col + iter_mv.col, reftex_offset_y + 0.5 + offset_mv.row + iter_mv.row ) * 255)      ) );
        sad_result  = __vabsdiff4( img0, ref0 );
		diff[i][TID]  = sad_result;
	}
	__syncthreads();

	// Accumulazione delle colonne di diff in modo da formare sad di blocchi per ogni candidato mv
	// Prima reduction, generazione 16 sad 4x4
	// 0   1   2   3   |   8   9  10  11  |  16  17  18  19  |  24  25  26  27  <- j
	// ^   ^   ^   ^   |   ^   ^   ^   ^  |   ^   ^   ^   ^  |   ^   ^   ^   ^
	// 4   5   6   7   |  12  13  14  15  |  20  21  22  23  |  28  29  30  31  <- j + 4
	for (i = 0; i < 16; i++) {
		j = i + (i / 4) * 4;
		diff[TID   ][j] += diff[TID   ][j+4];
		diff[TID+32][j] += diff[TID+32][j+4];
		diff[TID+64][j] += diff[TID+64][j+4];
		diff[TID+96][j] += diff[TID+96][j+4];
	}
	__syncthreads();

	// Seconda reduction, generazione 4 sad 8x8
	//       4        |       12        |        20        |        28        <- (8 * i) + 4
	//       ^        |        ^        |         ^        |         ^
	// 0   1   8   9  |  2   3  10  11  |  16  17  24  25  |  18  19  26  27  <- [j j+1 j+8 j+9]
	for (i = 0; i < 4; i++) {
		j = 2 * i + (i / 2) * 12;			// genera 0, 2, 16, 18 per i = 0 .. 3
		diff[TID   ][(8 * i) + 4] = diff[TID   ][j] + diff[TID   ][j + 1] + diff[TID   ][j + 8] + diff[TID   ][j + 9];
		diff[TID+32][(8 * i) + 4] = diff[TID+32][j] + diff[TID+32][j + 1] + diff[TID+32][j + 8] + diff[TID+32][j + 9];
		diff[TID+64][(8 * i) + 4] = diff[TID+64][j] + diff[TID+64][j + 1] + diff[TID+64][j + 8] + diff[TID+64][j + 9];
		diff[TID+96][(8 * i) + 4] = diff[TID+96][j] + diff[TID+96][j + 1] + diff[TID+96][j + 8] + diff[TID+96][j + 9];
	}
	__syncthreads();

	// Terza reduction (a), generazione 2 sad 8x16
	//         8x16
	//    22    |    30		<- 22 + (i * 8)
	//     ^    |     ^
	//   4  20  |  12  28
	for (i = 0; i < 2; i++) {
		j = 4 + (8 * i);				// genera 4, 12 per i = 0..1
		diff[TID   ][22 + (i * 8)] = diff[TID   ][j] + diff[TID   ][j + 16];
		diff[TID+32][22 + (i * 8)] = diff[TID+32][j] + diff[TID+32][j + 16];
		diff[TID+64][22 + (i * 8)] = diff[TID+64][j] + diff[TID+64][j + 16];
		diff[TID+96][22 + (i * 8)] = diff[TID+96][j] + diff[TID+96][j + 16];
	}
	//__syncthreads(); // potrebbe non servire!

	// Terza reduction (b), generazione 2 sad 16x8
	//       16x8
	//    6    |    14		<- 6*(i+1) + 2*i = 8 * i + 6
	//    ^    |     ^
	//  4  12  |  20  28	<- [j j+8]
	for (i = 0; i < 2; i++) {
		j = 4 + (16 * i);					// genera 4, 20 per i = 0..1
		diff[TID   ][8 * i + 6] = diff[TID   ][j] + diff[TID   ][j + 8];
		diff[TID+32][8 * i + 6] = diff[TID+32][j] + diff[TID+32][j + 8];
		diff[TID+64][8 * i + 6] = diff[TID+64][j] + diff[TID+64][j + 8];
		diff[TID+96][8 * i + 6] = diff[TID+96][j] + diff[TID+96][j + 8];
	}
	__syncthreads();

	// Quarta reduction, generazione 1 sad 16x16
	//    31
	//     ^
	//  6    14
	diff[TID   ][31] = diff[TID   ][6] + diff[TID   ][14];
	diff[TID+32][31] = diff[TID+32][6] + diff[TID+32][14];
	diff[TID+64][31] = diff[TID+64][6] + diff[TID+64][14];
	diff[TID+96][31] = diff[TID+96][6] + diff[TID+96][14];
	__syncthreads();


	// Ricerca del minimo di ogni colonna. A noi interessano 25 delle 32 colonne,
	// ma per non creare divergenza tra i thread eseguiamo la ricerca anche dove non serve
	minpos[TID] = 0;
	__syncthreads();

	// 32 thread, ognuno ricerca il minimo lungo una colonna
	for( i = 1; i < 128; i++ ){
		if ( diff[0][TID] > diff[i][TID] ) {
			diff[0][TID] = diff[i][TID];
			minpos[TID] = i;
		}
	}

#if HAVE_CUDA_MV_ITER
    if (fabsf(MV_16x12_lookup_tex[ minpos[31] ].row) + fabsf(MV_16x12_lookup_tex[ minpos[31] ].col) < 14)   // forse era < 12...
        break;
    iter_mv.col += MV_16x12_lookup_tex[ minpos[31] ].col * 1.75;
    iter_mv.row += MV_16x12_lookup_tex[ minpos[31] ].row * 1.75;
    maxloops -= 1;
    }
#endif

	// Salva mv 16x16
	// Questo potrebbe essere fatto meglio, conj 25 thread che lavorano contemporaneamente,
	// ma devo studiare come indicizzare l'accesso alla matrice globale. C'ho voglia?
	if ( TID == 31 ) {
		MVs_g[MBoffset].as_mv.row = (short)((MV_16x12_lookup_tex[ minpos[TID] ].row + iter_mv.row) * 8);
		MVs_g[MBoffset].as_mv.col = (short)((MV_16x12_lookup_tex[ minpos[TID] ].col + iter_mv.col) * 8);
	}
    if (split_on == SPLITMV_ON) {
    	// salva mv 4x4
    	if ( TID < 16 ) {
    		MVs_split_g[MBoffset*24 + TID].as_mv.row = (short)((MV_16x12_lookup_tex[ minpos[TID + (TID / 4) * 4] ].row + iter_mv.row) * 8);
    		MVs_split_g[MBoffset*24 + TID].as_mv.col = (short)((MV_16x12_lookup_tex[ minpos[TID + (TID / 4) * 4] ].col + iter_mv.col) * 8);
    	}
    	// salva mv 8x8
    	if ( TID < 4 ) {
    		MVs_split_g[MBoffset*24 + 16 + TID].as_mv.row = (short)((MV_16x12_lookup_tex[ minpos[8 * TID + 4] ].row + iter_mv.row) * 8);
    		MVs_split_g[MBoffset*24 + 16 + TID].as_mv.col = (short)((MV_16x12_lookup_tex[ minpos[8 * TID + 4] ].col + iter_mv.col) * 8);
    	}
    	// salva mv 8x16 e 16x8
    	if ( TID < 2 ) {
    		MVs_split_g[MBoffset*24 + 20 + TID].as_mv.row = (short)((MV_16x12_lookup_tex[ minpos[8 * TID + 22] ].row + iter_mv.row) * 8);
    		MVs_split_g[MBoffset*24 + 20 + TID].as_mv.col = (short)((MV_16x12_lookup_tex[ minpos[8 * TID + 22] ].col + iter_mv.col) * 8);
    		MVs_split_g[MBoffset*24 + 22 + TID].as_mv.row = (short)((MV_16x12_lookup_tex[ minpos[8 * TID +  6] ].row + iter_mv.row) * 8);
    		MVs_split_g[MBoffset*24 + 22 + TID].as_mv.col = (short)((MV_16x12_lookup_tex[ minpos[8 * TID +  6] ].col + iter_mv.col) * 8);
    	}
    }
    __syncthreads();
	///////////////////////////////////////////////////////////////////////////////////////////
	// STEP 2: pixel-scale Motion Vector Search

	// 1.
	// Ricerca di un MV per ogni blocco 4x4
	// 16 blocchi, 2 thread per blocco. Stesso schema per decidere TID => thread 0 e 4 fanno 1 blocco; 1 e 5 il secondo, ecc...
	// Risultati sad memorizzati in diff[i][TID] con 0 < i < 15
	// Questa volta non possiamo piu' sfruttare che refptr punti alla stesso indice, quindi posso
	// calcolare contemporaneamente ogni sad per tid e accumulare, ma posso sfruttare il
	// parallelismo tra mv dello stesso tipo: prima calcolo in parall tutte le 4x4, poi le 8x8, ecc...

    if (split_on == SPLITMV_ON) {

	// Update refpointer al miglior mv
	j = (TID % 4) + (TID / 8) * 8;	// Genera 0 1 2 3 0 1 2 3 8 9 10 11 8 9 10 11 16 17...
									// perche' TID 0 e 4 vengono traslati dello stesso mv corrispondente
									// a quello ora presente in colonna 0 di minpos
	reftex_offset_x += (MV_16x12_lookup_tex[minpos[j]].col + iter_mv.col);
	reftex_offset_y += (MV_16x12_lookup_tex[minpos[j]].row + iter_mv.row);

	for (i = 0; i < 93; i++) {
  		int32_t sad_result;
		MV_ref offset_mv = MV_lookup_refin_tex[i];
		/*uint32_t ref0 = (uint32_t) ( ( (uint8_t)(tex2D<float>( ref_tex, reftex_offset_x + 3.5 + offset_mv.col, reftex_offset_y + 0.5 + offset_mv.row ) * 255.0) << 24) |
									 ( (uint8_t)(tex2D<float>( ref_tex, reftex_offset_x + 2.5 + offset_mv.col, reftex_offset_y + 0.5 + offset_mv.row ) * 255.0) << 16) |
								     ( (uint8_t)(tex2D<float>( ref_tex, reftex_offset_x + 1.5 + offset_mv.col, reftex_offset_y + 0.5 + offset_mv.row ) * 255.0) <<  8) |
								     ( (uint8_t)(tex2D<float>( ref_tex, reftex_offset_x + 0.5 + offset_mv.col, reftex_offset_y + 0.5 + offset_mv.row ) * 255.0)      ) );
		uint32_t ref1 = (uint32_t) ( ( (uint8_t)(tex2D<float>( ref_tex, reftex_offset_x + 3.5 + offset_mv.col, reftex_offset_y + 1.5 + offset_mv.row ) * 255.0) << 24) |
									 ( (uint8_t)(tex2D<float>( ref_tex, reftex_offset_x + 2.5 + offset_mv.col, reftex_offset_y + 1.5 + offset_mv.row ) * 255.0) << 16) |
								     ( (uint8_t)(tex2D<float>( ref_tex, reftex_offset_x + 1.5 + offset_mv.col, reftex_offset_y + 1.5 + offset_mv.row ) * 255.0) <<  8) |
								     ( (uint8_t)(tex2D<float>( ref_tex, reftex_offset_x + 0.5 + offset_mv.col, reftex_offset_y + 1.5 + offset_mv.row ) * 255.0)      ) );
		sad_result  = __vabsdiff4( img0, ref0 );
		sad_result += __vabsdiff4( img1, ref1 );
		//sad_result  = __vvariance4( img0, ref0 );
		//sad_result += __vvariance4( img1, ref1 );
        */
        uint32_t ref0 = (uint32_t) ( ( (uint8_t)(tex2D<float>( ref_tex, reftex_offset_x + 3.5 + offset_mv.col, reftex_offset_y + 1.5 + offset_mv.row ) * 255.0) << 24) |
									 ( (uint8_t)(tex2D<float>( ref_tex, reftex_offset_x + 2.5 + offset_mv.col, reftex_offset_y + 0.5 + offset_mv.row ) * 255.0) << 16) |
								     ( (uint8_t)(tex2D<float>( ref_tex, reftex_offset_x + 1.5 + offset_mv.col, reftex_offset_y + 1.5 + offset_mv.row ) * 255.0) <<  8) |
								     ( (uint8_t)(tex2D<float>( ref_tex, reftex_offset_x + 0.5 + offset_mv.col, reftex_offset_y + 0.5 + offset_mv.row ) * 255.0)      ) );
        sad_result  = __vabsdiff4( img0, ref0 );
        diff[i][TID] = sad_result;
	}
	__syncthreads();

	for (i = 0; i < 16; i++) {
		j = i + (i / 4) * 4;
		diff[TID   ][j] += diff[TID   ][j+4];
		diff[TID+32][j] += diff[TID+32][j+4];
		diff[TID+64][j] += diff[TID+64][j+4];
	}

	minpos_refin[TID] = 0;
	__syncthreads();
	for( i = 1; i < 93; i++ ){
		if ( diff[0][TID] > diff[i][TID] ) {
			diff[0][TID] = diff[i][TID];
			minpos_refin[TID] = i;
		}
	}
	__syncthreads();

	// salva MV della split 4x4
	if ( TID < 16 ) {
		MVs_split_g[MBoffset*24 + TID].as_mv.row += (short)(MV_lookup_refin_tex[ minpos_refin[TID + (TID / 4) * 4] ].row * 8);
		MVs_split_g[MBoffset*24 + TID].as_mv.col += (short)(MV_lookup_refin_tex[ minpos_refin[TID + (TID / 4) * 4] ].col * 8);

	}

	// 2.
	// Ricerca di un mv per ogni blocco 8x8
	// Procedura esattamente identica alla precedente: TID che elaborano stesso blocco avranno
	// mv impostato coerentemente. Differente accumulazione (per blocco 0: TID 0 1 4 5 8 9 12 13)

	// Update refpointer al miglior mv
	//j = (TID / 8) * 8 + 4;	// Genera 4 4 4 4 4 4 4 4 12 12 12 12 12 12 12 12 20 20 20 20...
	j = 8 * ((TID / 2) % 2) + 4 + 16 * (TID / 16); // Genera 4 4 12 12 4 4 12 12 4 4 12 12 4 4 12 12 20 20 28 28 20 20 28 ecc..
	reftex_offset_x = saved_reftex_x + MV_16x12_lookup_tex[minpos[j]].col + iter_mv.col;
	reftex_offset_y = saved_reftex_y + MV_16x12_lookup_tex[minpos[j]].row + iter_mv.row;

	for (i = 0; i < 93; i++) {
  		int32_t sad_result;
		MV_ref offset_mv = MV_lookup_refin_tex[i];
        /*
		uint32_t ref0 = (uint32_t) ( ( (uint8_t)(tex2D<float>( ref_tex, reftex_offset_x + 3.5 + offset_mv.col, reftex_offset_y + 0.5 + offset_mv.row ) * 255.0) << 24) |
									 ( (uint8_t)(tex2D<float>( ref_tex, reftex_offset_x + 2.5 + offset_mv.col, reftex_offset_y + 0.5 + offset_mv.row ) * 255.0) << 16) |
								     ( (uint8_t)(tex2D<float>( ref_tex, reftex_offset_x + 1.5 + offset_mv.col, reftex_offset_y + 0.5 + offset_mv.row ) * 255.0) <<  8) |
								     ( (uint8_t)(tex2D<float>( ref_tex, reftex_offset_x + 0.5 + offset_mv.col, reftex_offset_y + 0.5 + offset_mv.row ) * 255.0)      ) );
		uint32_t ref1 = (uint32_t) ( ( (uint8_t)(tex2D<float>( ref_tex, reftex_offset_x + 3.5 + offset_mv.col, reftex_offset_y + 1.5 + offset_mv.row ) * 255.0) << 24) |
									 ( (uint8_t)(tex2D<float>( ref_tex, reftex_offset_x + 2.5 + offset_mv.col, reftex_offset_y + 1.5 + offset_mv.row ) * 255.0) << 16) |
								     ( (uint8_t)(tex2D<float>( ref_tex, reftex_offset_x + 1.5 + offset_mv.col, reftex_offset_y + 1.5 + offset_mv.row ) * 255.0) <<  8) |
								     ( (uint8_t)(tex2D<float>( ref_tex, reftex_offset_x + 0.5 + offset_mv.col, reftex_offset_y + 1.5 + offset_mv.row ) * 255.0)      ) );
		sad_result  = __vabsdiff4( img0, ref0 );
		sad_result += __vabsdiff4( img1, ref1 );
        */
	    //sad_result  = __vvariance4( img0, ref0 );
		//sad_result += __vvariance4( img1, ref1 );
        uint32_t ref0 = (uint32_t) ( ( (uint8_t)(tex2D<float>( ref_tex, reftex_offset_x + 3.5 + offset_mv.col, reftex_offset_y + 1.5 + offset_mv.row ) * 255.0) << 24) |
									 ( (uint8_t)(tex2D<float>( ref_tex, reftex_offset_x + 2.5 + offset_mv.col, reftex_offset_y + 0.5 + offset_mv.row ) * 255.0) << 16) |
								     ( (uint8_t)(tex2D<float>( ref_tex, reftex_offset_x + 1.5 + offset_mv.col, reftex_offset_y + 1.5 + offset_mv.row ) * 255.0) <<  8) |
								     ( (uint8_t)(tex2D<float>( ref_tex, reftex_offset_x + 0.5 + offset_mv.col, reftex_offset_y + 0.5 + offset_mv.row ) * 255.0)      ) );
        sad_result  = __vabsdiff4( img0, ref0 );
        diff[i][TID] = sad_result;
	}
	__syncthreads();

	// Sono pigro, copio e incollo la stessa manfrina
	for (i = 0; i < 16; i++) {
		j = i + (i / 4) * 4;
		diff[TID   ][j] += diff[TID   ][j+4];
		diff[TID+32][j] += diff[TID+32][j+4];
		diff[TID+64][j] += diff[TID+64][j+4];
	}
	__syncthreads();
	for (i = 0; i < 4; i++) {
		j = 2 * i + (i / 2) * 12;
		diff[TID   ][(8 * i) + 4] = diff[TID   ][j] + diff[TID   ][j + 1] + diff[TID   ][j + 8] + diff[TID]   [j + 9];
		diff[TID+32][(8 * i) + 4] = diff[TID+32][j] + diff[TID+32][j + 1] + diff[TID+32][j + 8] + diff[TID+32][j + 9];
		diff[TID+64][(8 * i) + 4] = diff[TID+64][j] + diff[TID+64][j + 1] + diff[TID+64][j + 8] + diff[TID+64][j + 9];
	}
	__syncthreads();

	minpos_refin[TID] = 0;
	__syncthreads();
	// 32 thread, ognuno ricerca il minimo lungo ogni colonna
	// anche se le colonne interessanti sono solo la 4, 12, 20 e 28
	for( i = 1; i < 93; i++ ){
		if ( diff[0][TID] > diff[i][TID] ) {
			diff[0][TID] = diff[i][TID];
			minpos_refin[TID] = i;
		}
	}
	__syncthreads();

	// Salva i MV della split 8x8
	if ( TID < 4 ) {
		MVs_split_g[MBoffset*24 + 16 + TID].as_mv.row += (short)(MV_lookup_refin_tex[ minpos_refin[8 * TID + 4] ].row * 8);
		MVs_split_g[MBoffset*24 + 16 + TID].as_mv.col += (short)(MV_lookup_refin_tex[ minpos_refin[8 * TID + 4] ].col * 8);
	}

	// 4.
	// Ricerca di un mv per ogni blocco 8x16
	// TODO?
	// Ho paura che diventi un'operazione un po' troppo onerosa, per ora preferisco lasciare
	// le 8x16 con dettaglio a due pixel (senza refining search)

    }

	// 5.
	// Refining search su blocco 16x16
	// Update RefPointer to the best motion vector
	reftex_offset_x = saved_reftex_x + (MV_16x12_lookup_tex[minpos[31]]).col + iter_mv.col;
	reftex_offset_y = saved_reftex_y + (MV_16x12_lookup_tex[minpos[31]]).row + iter_mv.row;

	for (i = 0; i < 93; i++) {
  		int32_t sad_result;
		MV_ref offset_mv = MV_lookup_refin_tex[i];
        /*
		uint32_t ref0 = (uint32_t) ( ( (uint8_t)(tex2D<float>( ref_tex, reftex_offset_x + 3.5 + offset_mv.col, reftex_offset_y + 0.5 + offset_mv.row ) * 255.0) << 24) |
									 ( (uint8_t)(tex2D<float>( ref_tex, reftex_offset_x + 2.5 + offset_mv.col, reftex_offset_y + 0.5 + offset_mv.row ) * 255.0) << 16) |
								     ( (uint8_t)(tex2D<float>( ref_tex, reftex_offset_x + 1.5 + offset_mv.col, reftex_offset_y + 0.5 + offset_mv.row ) * 255.0) <<  8) |
								     ( (uint8_t)(tex2D<float>( ref_tex, reftex_offset_x + 0.5 + offset_mv.col, reftex_offset_y + 0.5 + offset_mv.row ) * 255.0)      ) );
		uint32_t ref1 = (uint32_t) ( ( (uint8_t)(tex2D<float>( ref_tex, reftex_offset_x + 3.5 + offset_mv.col, reftex_offset_y + 1.5 + offset_mv.row ) * 255.0) << 24) |
									 ( (uint8_t)(tex2D<float>( ref_tex, reftex_offset_x + 2.5 + offset_mv.col, reftex_offset_y + 1.5 + offset_mv.row ) * 255.0) << 16) |
								     ( (uint8_t)(tex2D<float>( ref_tex, reftex_offset_x + 1.5 + offset_mv.col, reftex_offset_y + 1.5 + offset_mv.row ) * 255.0) <<  8) |
								     ( (uint8_t)(tex2D<float>( ref_tex, reftex_offset_x + 0.5 + offset_mv.col, reftex_offset_y + 1.5 + offset_mv.row ) * 255.0)      ) );
		sad_result  = __vabsdiff4( img0, ref0 );
		sad_result += __vabsdiff4( img1, ref1 );
		//sad_result  = __vvariance4( img0, ref0 );
		//sad_result += __vvariance4( img1, ref1 );
        */
        uint32_t ref0 = (uint32_t) ( ( (uint8_t)(tex2D<float>( ref_tex, reftex_offset_x + 3.5 + offset_mv.col, reftex_offset_y + 1.5 + offset_mv.row ) * 255.0) << 24) |
									 ( (uint8_t)(tex2D<float>( ref_tex, reftex_offset_x + 2.5 + offset_mv.col, reftex_offset_y + 0.5 + offset_mv.row ) * 255.0) << 16) |
								     ( (uint8_t)(tex2D<float>( ref_tex, reftex_offset_x + 1.5 + offset_mv.col, reftex_offset_y + 1.5 + offset_mv.row ) * 255.0) <<  8) |
								     ( (uint8_t)(tex2D<float>( ref_tex, reftex_offset_x + 0.5 + offset_mv.col, reftex_offset_y + 0.5 + offset_mv.row ) * 255.0)      ) );
        sad_result  = __vabsdiff4( img0, ref0 );
        diff[i][TID] = sad_result;
	}
	__syncthreads();

	for (i=0; i<16; i++) {
		diff[TID   ][i] += diff[TID   ][i+16];
		diff[TID+32][i] += diff[TID+32][i+16];
		diff[TID+64][i] += diff[TID+64][i+16];
	}
	__syncthreads();
	for (i=0; i<8; i++) {
		diff[TID   ][i] += diff[TID   ][i+8];
		diff[TID+32][i] += diff[TID+32][i+8];
		diff[TID+64][i] += diff[TID+64][i+8];
	}
	__syncthreads();
	for (i=0; i<4; i++) {
		diff[TID   ][i] += diff[TID   ][i+4];
		diff[TID+32][i] += diff[TID+32][i+4];
		diff[TID+64][i] += diff[TID+64][i+4];
	}
	__syncthreads();
	diff[TID   ][0]    += (diff[TID   ][1] + diff[TID   ][2] + diff[TID   ][3]);
	diff[TID+32][0]    += (diff[TID+32][1] + diff[TID+32][2] + diff[TID+32][3]);
	diff[TID+64][0]    += (diff[TID+64][1] + diff[TID+64][2] + diff[TID+64][3]);
	__syncthreads();

	minpos_refin[TID] = 0;
	__syncthreads();

	// 32 thread, ognuno ricerca il minimo lungo ogni colonna
	// anche se le colonne interessanti sono solo la 0
	for( i = 1; i < 93; i++ ){
		if ( diff[0][TID] > diff[i][TID] ) {
			diff[0][TID] = diff[i][TID];
			minpos_refin[TID] = i;
		}
	}
	__syncthreads();

    if( TID == 0 )
	{
		MVs_g[MBoffset].as_mv.row += (short)(MV_lookup_refin_tex[ minpos_refin[0] ].row * 8);
		MVs_g[MBoffset].as_mv.col += (short)(MV_lookup_refin_tex[ minpos_refin[0] ].col * 8);
    }

}


inline void me_kernel_launch_tex( VP8_COMMON * const common, const hipTextureObject_t in_tex, const hipTextureObject_t ref_tex,
		int const streamID, int const split_on, int_mv * const MVs, int_mv * const MVs_split ) {

#if CUDA_VERBOSE
	float elapsedTime;
	hipEvent_t start, stop;
	CHECK(hipEventCreate(&start));
	CHECK(hipEventCreate(&stop));
	CHECK(hipEventRecord(start));
#endif

	me_cuda_tex <<< common->GPU.gridDim, common->GPU.blockDim, 0, common->GPU.streams.frame[streamID] >>> (in_tex, ref_tex,
			streamID, common->GPU.streamSize, common->gpu_frame.stride, common->gpu_frame.width, common->gpu_frame.num_MB_width, split_on, MVs, MVs_split );

#if CUDA_VERBOSE
	CHECK(hipEventRecord(stop));
	CHECK(hipEventSynchronize(stop));
	CHECK(hipEventElapsedTime(&elapsedTime, start, stop));
	printf("\n[GPU] ME elapsed time streams[%d]:  %.4f ms\n",streamID,elapsedTime);
	CHECK(hipEventDestroy(start));
	CHECK(hipEventDestroy(stop));
	add_STATS((double)elapsedTime,0);
#endif

}

void me_cuda_launch_interleaved_tex( VP8_COMMON * const cm, int fb_idx, int ref_frame_flags ) {

	//int MV_size_16 = 16*sizeof(int_mv);
	int MV_size_16 = cm->GPU.streamSize * sizeof(int_mv);
	// for printing informations about reference frame flags and their usage, I left a commented prinft at line 3625
	// at the beginning of encode_frame_to_data_rate(..) in onyx_if.c

	for (int t = 0; t < cm->GPU.num_mb16th; t++) {

		int s = cm->GPU.streamLaunchOrder[t];
		//int offset = 16*s;
		int offset = cm->GPU.streamSize * s;
		// bugfix per immagini il cui n di mb non e' divisibile per 16
		// prima venivano lanciati troppi processi e hipMemcpyAsync andava a leggere oltre i limiti degli array
		if (offset + cm->GPU.streamSize > cm->gpu_frame.num_mv)
			MV_size_16 = ( offset + cm->GPU.streamSize - cm->gpu_frame.num_mv ) * sizeof( int_mv );


		if ((ref_frame_flags & GPUFLAG_LAST_FRAME) && (cm->yv12_fb[cm->lst_fb_idx].flags & GPUFLAG_LAST_FRAME)) {
            me_kernel_launch_tex(cm, cm->gpu_frame.rawFbTex, (cm->gpu_frame.fbTex)[cm->lst_fb_idx], s, SPLITMV_ON, (cm->gpu_frame.MVs_g)[0], (cm->gpu_frame.MVs_split_g) );
			CHECK(hipMemcpyAsync( &(cm->host_frame.MVs_h)[0][offset],		&(cm->gpu_frame.MVs_g)[0][offset],		MV_size_16,      hipMemcpyDeviceToHost, cm->GPU.streams.frame[s]));
			CHECK(hipMemcpyAsync( &(cm->host_frame.MVs_split_h)[offset],&(cm->gpu_frame.MVs_split_g)[offset],24 * MV_size_16, hipMemcpyDeviceToHost, cm->GPU.streams.frame[s]));
		}

		// Se ref_frame_flags indica la presenza di un gold e se il flag del fb puntato da gld_fb_idx indica che e' gold, allora...
		if ((ref_frame_flags & GPUFLAG_GOLD_FRAME) && (cm->yv12_fb[cm->gld_fb_idx].flags & GPUFLAG_GOLD_FRAME)) {
			me_kernel_launch_tex(cm, cm->gpu_frame.rawFbTex, (cm->gpu_frame.fbTex)[cm->gld_fb_idx], s, SPLITMV_OFF, (cm->gpu_frame.MVs_g)[1], 0 );
			CHECK(hipMemcpyAsync( &(cm->host_frame.MVs_h)[1][offset],		&(cm->gpu_frame.MVs_g)[1][offset],		MV_size_16,      hipMemcpyDeviceToHost, cm->GPU.streams.frame[s]));
			//CHECK(hipMemcpyAsync( &(cm->host_frame.MVs_split_h)[1][offset],&(cm->gpu_frame.MVs_split_g)[1][offset],24 * MV_size_16, hipMemcpyDeviceToHost, cm->GPU.streams.frame[s]));
		}

		// Se ref_frame_flags indica la presenza di un altref e se il flag del fb puntato da alt_fb_idx indica che e' altref, allora...
		if ((ref_frame_flags & GPUFLAG_ALTR_FRAME) && (cm->yv12_fb[cm->alt_fb_idx].flags & GPUFLAG_ALTR_FRAME)) {
			me_kernel_launch_tex(cm, cm->gpu_frame.rawFbTex, (cm->gpu_frame.fbTex)[cm->alt_fb_idx], s, SPLITMV_OFF, (cm->gpu_frame.MVs_g)[2], 0 );
			CHECK(hipMemcpyAsync( &(cm->host_frame.MVs_h)[2][offset],		&(cm->gpu_frame.MVs_g)[2][offset],		MV_size_16,		 hipMemcpyDeviceToHost, cm->GPU.streams.frame[s]));
			//CHECK(hipMemcpyAsync( &(cm->host_frame.MVs_split_h)[2][offset],&(cm->gpu_frame.MVs_split_g)[2][offset],24 * MV_size_16, hipMemcpyDeviceToHost, cm->GPU.streams.frame[s]));
		}
	}
}


#endif  /* HAVE_CUDA_ENABLED_DEVICE */
#ifdef __cplusplus
}
#endif
